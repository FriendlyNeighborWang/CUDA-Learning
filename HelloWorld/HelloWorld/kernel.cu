#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>

// __global__标识符意味着该函数是可以由主机调用的设备函数
__global__ void add(int a, int b, int* c) {
	*c = a + b;
}

// 下面这段主要是将数据搬到了GPU上
int main(void) {
	int c;
	int* dev_c;
	// 在设备上分配内存
	hipMalloc((void**)&dev_c, sizeof(int));
	add << <1, 1 >> > (2, 7, dev_c);
	// 将主机内存上的数据搬到GPU显存上
	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("2+7 = %d\n", c);
	hipFree(dev_c);
}